#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

//#define DEBUG
#define TYPE double
#define BLOCK_SIZE 64
#define MAX_SIZE 4000
TYPE *a; // matriz de coeficientes
TYPE *x;
TYPE *b;
TYPE *swap;

//
__global__ void gElim(TYPE* a, TYPE* b, int n, int i) {

  int row = (i+1) + blockIdx.x * blockDim.x + threadIdx.x;
  if (row < n ) {
		TYPE m = a[row*n + i] / a[i*n+i];
		for (int k = i; k < n; k++) {
			a[row*n + k] = a[row*n + k] - m * a[i*n+k];
		}
		b[row] = b[row] - m * b[i];
  }
}

int main() {
	int r, n;
	int i, j;
	TYPE m;

	FILE *file;

	if ((file = fopen("gauss.in", "r")) == NULL) {
		perror("gauss.in");
		exit(1);
	}

	if(!fscanf(file, "%d", &n)){
		perror("n in gauss.in");
		exit(1);
	}

	a = (TYPE*) malloc(n*n*sizeof(TYPE));
	x = (TYPE*) malloc(n*sizeof(TYPE));
	b = (TYPE*) malloc(n*sizeof(TYPE));
	swap = (TYPE*) malloc(n*sizeof(TYPE));

	for (r = 0; r < n * n; r++) {
		if(!fscanf(file, "%lf", &a[r])){
			perror("a in gauss.in");
			exit(1);
		}
	}

	for (r = 0; r < n; r++){
		if(!fscanf(file, "%lf", &b[r])){
			perror("b in gauss.in");
			exit(1);
		}
	}

	fclose(file);

	hipEvent_t start, stop;

	// Criacao dos eventos
	hipEventCreate(&start);
	hipEventCreate(&stop);

  TYPE *d_a, *d_b;
  hipMalloc((void **)&d_a, n*n*sizeof(TYPE));
  hipMalloc((void **)&d_b, n*sizeof(TYPE));

  int block = BLOCK_SIZE;
	int grid = (n + block - 1) / block;

	hipEventRecord(start);

	hipMemcpy(d_a, a, n*n*sizeof(TYPE), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n*sizeof(TYPE), hipMemcpyHostToDevice);


	for (i = 0; i < n - 1; i++) {

		gElim <<< grid, block >>> (d_a, d_b, n, i);
		hipError_t erro = hipGetLastError();
		if(erro != hipSuccess){
			printf("Erro na ativação do kernel: %s\n", hipGetErrorString(erro));
			exit(1);
		}
	}
	hipMemcpy(a, d_a, n*n*sizeof(TYPE), hipMemcpyDeviceToHost);
  hipMemcpy(b, d_b, n*sizeof(TYPE), hipMemcpyDeviceToHost);

	hipEventRecord(stop);

	x[i] = b[i] / a[i*n + i];
	for (i = i - 1; i >= 0; i--) {
		m = 0;
		for (j = i; j < n; j++)
			m = m + a[i*n + j] * x[j];
		x[i] = (b[i] - m) / a[i*n + i];
	}

	// garante que o evento stop já ocorreu
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Tempo de Execução na GPU: %.4f s\n", milliseconds/1000);

//#define DEBUG
#ifdef DEBUG
	for (int r = 0; r < n; r++) {
		for (int c = 0; c < n; c++) {
			fprintf(stdout, "%.2lf ", a[r*n + c]);
		}
		fprintf(stdout, "\n");
	}
  fprintf(stdout, "\n");
	for (r = 0; r < n; r++)
	fprintf(stdout, "%.2lf ", b[r]);
	fprintf(stdout, "\n");
#endif

	if ((file = fopen("gaussCUDA.out", "w")) == NULL) {
		perror("gaussCUDA.out");
		exit(1);
	}

	for (r = 0; r < n; r++)
		fprintf(file, "%2.2lf ", x[r]);
	fprintf(file, "\n");
	for (r = 0; r < 10; r++)
		fprintf(stdout, "%2.2lf ", x[r]);
	fprintf(stdout, "\n");
	fflush(file);
	fclose(file);

	free(a);
  free(x);
  free(b);
  free(swap);
  hipFree(d_a);
  hipFree(d_b);

	return EXIT_SUCCESS;
}